#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 
 #include "WindowsMaskPlugin.h"

using namespace nvinfer1;

PluginFieldCollection WindowsMaskPluginCreator::fc_{};
std::vector<PluginField> WindowsMaskPluginCreator::attr_;

__global__ void windowsMaskKernel(float *pInput, int *shape, float *pOutput, int nElement)
{
    const int index = blockIdx.x * 64 + threadIdx.x;
    if (index > nElement){
        return;
    }
    int H = shape[1];
    int W = shape[2];

    int h = index / W;
    int w = index % W;
    
    if (h < (H - 8) && w < (W - 8)){
        pOutput[index] = 0;
    }
    else if (h < (H - 8) && w < (W - 4) && w >= (W - 8)){
        pOutput[index] = 1;
    }
    else if (h < (H - 8) && w >= (W - 4)){
        pOutput[index] = 2;
    }
    else if (h < (H - 4) && h >= (H - 8) && w < (W - 8)){
        pOutput[index] = 3;
    }
    else if (h < (H - 4) && h >= (H - 8) && w < (W - 4) && w >= (W - 8)){
        pOutput[index] = 4;
    }
    else if (h < (H - 4) && h >= (H - 8) && w >= (W - 4)){
        pOutput[index] = 5;
    }
    else if (h >= (H - 4) && w < (W - 8)){
        pOutput[index] = 6;
    }
    else if (h >= (H - 4) && w < (W - 4) && w >= (W - 8)){
        pOutput[index] = 7;
    }
    else if (h >= (H - 4) && w >= (W - 4)){
        pOutput[index] = 8;
    }
}

int32_t WindowsMaskPlugin::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{
    int nElement = 1;
    for (int i = 0; i < inputDesc[0].dims.nbDims; i++)
    {
        nElement *= inputDesc[0].dims.d[i];
    }

    dim3 grid(CEIL_DIVIDE(nElement, 64), 1, 1), block(64, 1, 1); 
    windowsMaskKernel <<<grid, block, 0, stream>>>((float *)inputs[0], (int *)inputs[1], (float *)outputs[0], nElement);
    return 0;
}

REGISTER_TENSORRT_PLUGIN(WindowsMaskPluginCreator);

