#include "hip/hip_runtime.h"
#include "RollPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    RollPluginCreator::fc_ {};
std::vector<PluginField> RollPluginCreator::attr_;

__global__ void RollVerticalKernel(float *pInput, int fea_b, int fea_h, int fea_w, int fea_c, int shift, float *pOutput)
{
    // B, H, W, C
    const int index = blockIdx.x * 256 + threadIdx.x;
    int h = index / (fea_w * fea_c) % fea_h;
    int b = index / (fea_h * fea_w * fea_c);
    int target_h = (fea_h + shift + h) % fea_h;
    int target_pos = b * (fea_h * fea_w * fea_c) + target_h * (fea_w * fea_c) + index % (fea_w * fea_c);
    pOutput[target_pos] = pInput[index];
}

__global__ void RollHorizontalKernel(float *pInput, int fea_b, int fea_h, int fea_w, int fea_c, int shift, float *pOutput)
{
    const int index = blockIdx.x * 256 + threadIdx.x;
    int w = index / fea_c % fea_w;
    int b_h = index / (fea_w * fea_c);
    int target_w = (fea_w + shift + w) % fea_w;
    int target_pos = b_h * (fea_w * fea_c) + target_w * fea_c + index % fea_c;
    pOutput[target_pos] = pInput[index];
}

int32_t RollPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int fea_b = inputDesc[0].dims.d[0], fea_h = inputDesc[0].dims.d[1], fea_w = inputDesc[0].dims.d[2], fea_c = inputDesc[0].dims.d[3];
    int nElement = 1;
    for (int i = 0; i < inputDesc[0].dims.nbDims; i++)
    {
        nElement *= inputDesc[0].dims.d[i];
    }

    dim3 grid(CEIL_DIVIDE(nElement, 256), 1, 1), block(256, 1, 1); 
    if (m.direction_ == 0){
        RollVerticalKernel<<<grid, block, 0, stream>>>((float *)inputs[0], fea_b, fea_h, fea_w, fea_c, m.shift_, (float *)outputs[0]);
    }
    else if (m.direction_ == 1){
        RollHorizontalKernel<<<grid, block, 0, stream>>>((float *)inputs[0], fea_b, fea_h, fea_w, fea_c, m.shift_, (float *)outputs[0]);
    }
    else{
        printf("No implement!");
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(RollPluginCreator);