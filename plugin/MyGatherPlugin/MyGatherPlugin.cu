#include "hip/hip_runtime.h"
#include "MyGatherPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    MyGatherPluginCreator::fc_ {};
std::vector<PluginField> MyGatherPluginCreator::attr_;

template<typename T>
__global__ void MyGatherKernel(T *pInput, T *pScale, int nfea, T *pOutput0, T *pOutput1, T *pOutput2)
{
    const int index = blockIdx.x * 256 + threadIdx.x;
    int target_pos = index % nfea;
    int target_dim = index / nfea;
    if (target_dim == 0){
        pOutput0[target_pos] = pInput[index] * pScale[0];
    }
    else if (target_dim == 1){
        pOutput1[target_pos] = pInput[index];
    }
    else{
        pOutput2[target_pos] = pInput[index];
    }
}

int32_t MyGatherPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int nElement = 1;
    for (int i = 0; i < inputDesc[0].dims.nbDims; i++)
    {
        nElement *= inputDesc[0].dims.d[i];
    }
    int nfea = 1;
    for (int i = 1; i < inputDesc[0].dims.nbDims; i++)
    {
        nfea *= inputDesc[0].dims.d[i];
    }

    dim3 grid(CEIL_DIVIDE(nElement, 256), 1, 1), block(256, 1, 1);
    switch (int(inputDesc[0].type)){
        case int(DataType::kFLOAT):{
            MyGatherKernel<float><<<grid, block, 0, stream>>>((float *)inputs[0], (float *)inputs[1], nfea, (float *)outputs[0], (float *)outputs[1], (float *)outputs[2]);
            break;
        }
        case int(DataType::kHALF):{
            MyGatherKernel<half><<<grid, block, 0, stream>>>((half *)inputs[0], (half *)inputs[1], nfea, (half *)outputs[0], (half *)outputs[1], (half *)outputs[2]);
            break;
        }
        default:
            printf("DataType not support!\n");
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(MyGatherPluginCreator);