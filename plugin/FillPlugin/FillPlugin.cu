#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 
 #include "FillPlugin.h"

using namespace nvinfer1;

PluginFieldCollection FillPluginCreator::fc_{};
std::vector<PluginField> FillPluginCreator::attr_;

template<typename T>
__global__ void fillKernel(T *pInput, T *pOutput)
{
    const int tx = threadIdx.x, index = blockIdx.x * 256 + threadIdx.x;

    if (pInput[index] > (T)0.1 || pInput[index] < (T)-0.1){
        pOutput[index] = (T)-100.0;
    }
    else{
        pOutput[index] = 0.0;
    }
}

int32_t FillPlugin::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{
    int nElement = 1;
    for (int i = 0; i < inputDesc[0].dims.nbDims; i++)
    {
        nElement *= inputDesc[0].dims.d[i];
    }

    dim3 grid(CEIL_DIVIDE(nElement, 256), 1, 1), block(256, 1, 1);
    switch (int(inputDesc[0].type)){
        case int(DataType::kFLOAT):{
            fillKernel<float> <<<grid, block, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);
            break;
        }
        case int(DataType::kHALF):{
            fillKernel<half> <<<grid, block, 0, stream>>>((half *)inputs[0], (half *)outputs[0]);
            break;
        }
        default:
            printf("DataType not support!\n");
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(FillPluginCreator);

