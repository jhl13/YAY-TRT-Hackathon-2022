#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 
 #include "FillPlugin.h"

using namespace nvinfer1;

PluginFieldCollection FillPluginCreator::fc_{};
std::vector<PluginField> FillPluginCreator::attr_;

__global__ void fillKernel(float *pInput, float *pOutput)
{
    const int tx = threadIdx.x, index = blockIdx.x * 256 + threadIdx.x;

    if (pInput[index] != 0){
        pOutput[index] = -100.0;
    }
}

int32_t FillPlugin::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{
    int nElement = 1;
    for (int i = 0; i < inputDesc[0].dims.nbDims; i++)
    {
        nElement *= inputDesc[0].dims.d[i];
    }

    dim3 grid(CEIL_DIVIDE(nElement, 256), 1, 1), block(256, 1, 1);
    fillKernel <<<grid, block, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);
    return 0;
}

REGISTER_TENSORRT_PLUGIN(FillPluginCreator);

